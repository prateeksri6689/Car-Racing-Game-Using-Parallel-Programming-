#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <math.h>

#define N 100000
using namespace std;
static const long BLK_SIZE =1000 ;
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }



__global__ void sort(int *c,int *count)
{
    int l;
    if(*count%2==0)
          l=*count/2;
    else
         l=(*count/2)+1;
    for(int i=0;i<l;i++)
    {
            if(threadIdx.x%2==0)  //even phase
            {
                if(c[threadIdx.x]>c[threadIdx.x+1])
                {
                    int temp=c[threadIdx.x];
                    c[threadIdx.x]=c[threadIdx.x+1];
                    c[threadIdx.x+1]=temp;
                }

            __syncthreads();
            }
            else     //odd phase
            {
                if(c[threadIdx.x]>c[threadIdx.x+1])
                {
                    int temp=c[threadIdx.x];
                    c[threadIdx.x]=c[threadIdx.x+1];
                    c[threadIdx.x+1]=temp;
                }

            __syncthreads();
            }
    }

}

void swap(int *xp, int *yp)
{
	int temp = *xp;
	*xp = *yp;
	*yp = temp;
}

// An optimized version of Bubble Sort
void bubbleSort(int arr[], int n)
{

}

int main()
{
int a[N],b[N];
    for (int i = 0; i < N; i++) {
  		a[i] = (float) rand() / (float) RAND_MAX * 100;

  	}


  printf("ORIGINAL ARRAY : \n");
  for(int i=0;i<N;i++)
          {

          printf("%d ",a[i]);
          }


  int *c,*count;
  int k=N;


  hipMalloc((void**)&c,sizeof(int)*N);
  hipMalloc((void**)&count,sizeof(int));
  hipMemcpy(c,&a,sizeof(int)*N,hipMemcpyHostToDevice);
  hipMemcpy(count,&k,sizeof(int),hipMemcpyHostToDevice);


  //Time kernel launch
  	//Time kernel launch
  	hipEvent_t start, stop;
  	CUDA_CHECK_RETURN(hipEventCreate(&start));
  	CUDA_CHECK_RETURN(hipEventCreate(&stop));
  	float elapsedTime;

  	CUDA_CHECK_RETURN(hipEventRecord(start, 0));



  sort<<< ceil(N/(float)BLK_SIZE),BLK_SIZE >>>(c,count);

  CUDA_CHECK_RETURN(hipEventRecord(stop, 0));

  	CUDA_CHECK_RETURN(hipEventSynchronize(stop));
  	CUDA_CHECK_RETURN(hipEventElapsedTime(&elapsedTime, start, stop));
  	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
  	CUDA_CHECK_RETURN(hipGetLastError()); //Check if an error occurred in device code
  	CUDA_CHECK_RETURN(hipEventDestroy(start));
  	CUDA_CHECK_RETURN(hipEventDestroy(stop));
  	cout << "done.\nElapsed kernel time: " << elapsedTime << " ms\n";

  	cout << "Copying results back to host .... "<<endl;

  hipMemcpy(&b,c,sizeof(int)*N,hipMemcpyDeviceToHost);
  printf("\nSORTED ARRAY : \n");

  for(int i=0;i<N;i++)
      {
         printf("%d ",b[i]);
      }

  //Add code to time host calculations
  	clock_t st, ed;

  	st = clock();
  	//bool valid = true;

  //bubbleSort(a,N);

  	int i, j;
  	bool swapped;
  	for (i = 0; i < N-1; i++)
  	{
  		swapped = false;
  		for (j = 0; j < N-i-1; j++)
  		{
  			if (a[j] > a[j+1])
  			{
  			swap(&a[j], &a[j+1]);
  			swapped = true;
  			}
  		}

  		// IF no two elements were swapped by inner loop, then break
  		if (swapped == false)
  			break;
  	}

  printf("\n");
        printf("BYCPU");
        printf("\n");
  for(int i=0;i<N;i++)
        {

	  printf("%d ",a[i]);
        }
  ed = clock() - st;
  	cout << "Elapsed time on host: " << ((float) ed) / CLOCKS_PER_SEC * 1000
  			<< " ms" << endl;

}



